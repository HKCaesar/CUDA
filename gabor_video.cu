#include "hip/hip_runtime.h"
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <iostream>
#include <opencv2/imgproc/imgproc.hpp>
#include <math.h>
#include <vector>

using namespace std;
using namespace cv;

cv::Mat seg;
cv::Mat src;
cv::Mat grey;
cv::Mat dest[12];


 cv::Mat mkKernel(int ks, double sig, double th, double lm, double ps)
      {
          int hks = (ks-1)/2;
          double theta = th*CV_PI/180;
          double psi = ps*CV_PI/180;
          double del = 2.0/(ks-1);
          double lmbd = lm;
          double sigma = sig/ks;
          double x_theta;
          double y_theta;
          cv::Mat kernel(ks,ks, CV_32F);
          for (int y=-hks; y<=hks; y++)
          {
              for (int x=-hks; x<=hks; x++)
              {
                  x_theta = x*del*cos(theta)+y*del*sin(theta);
                  y_theta = -x*del*sin(theta)+y*del*cos(theta);
                  kernel.at<float>(hks+y,hks+x) = (float)exp(-0.5*(pow(x_theta,2)+pow(y_theta,2))/pow(sigma,2))* cos(2*CV_PI*x_theta/lmbd + psi);
              }
          }
          return kernel;
      }

 int Process(int ks,int sigma, int lamda,  int psi)
     {

     	 seg=abs(grey);

     	for (int i=0;i<12;i++)
     	{
     		cv::Mat kernel = mkKernel(ks, sigma, i*30, lamda, psi);
     		cv::filter2D(grey, dest[i], CV_32F, kernel);
     		seg+=abs(dest[i]);



     	}

     	seg= seg-abs(grey);
     	seg/=4;

     	cv::imshow("seg",seg);
     	seg*=0;

         return 0;
     }


int main()
{

CvCapture *camera=cvCaptureFromFile("sample6.avi");
cvNamedWindow("img");
while (cvWaitKey(10)!=atoi("q"))
{
    double t1=(double)cvGetTickCount();
    IplImage *img=cvQueryFrame(camera);
    cv::Mat B1 = cvarrToMat(img);
    //IplImage * image = &((IplImage)B1);
    cvShowImage("img",img);
    //cv::imshow("img1",B1);
    cv::cvtColor(B1, src, CV_BGR2GRAY);
    //cv::imshow("img1",src);
    src.convertTo(grey, CV_32F, 1.0/255, 0);
    cv::imshow("img1",grey);
    Process(21, 5, 50,90);
}
cvReleaseCapture(&camera);
}
