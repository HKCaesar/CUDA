#include "hip/hip_runtime.h"
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <iostream>
#include <opencv2/imgproc/imgproc.hpp>
#include <math.h>
#include <vector>
#include <iostream>
#include <opencv2/opencv.hpp>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <float.h>
#include <limits.h>
#include <time.h>
#include <ctype.h>

using namespace std;
using namespace cv;

const char* cascade_name ="cars3.xml";
cv::Mat seg;
cv::Mat src;
cv::Mat grey;
cv::Mat dest[12];
IplImage *mark;
IplImage *img1;
 cv::Mat mkKernel(int ks, double sig, double th, double lm, double ps)
      {
          int hks = (ks-1)/2;
          double theta = th*CV_PI/180;
          double psi = ps*CV_PI/180;
          double del = 2.0/(ks-1);
          double lmbd = lm;
          double sigma = sig/ks;
          double x_theta;
          double y_theta;
          cv::Mat kernel(ks,ks, CV_32F);
          for (int y=-hks; y<=hks; y++)
          {
              for (int x=-hks; x<=hks; x++)
              {
                  x_theta = x*del*cos(theta)+y*del*sin(theta);
                  y_theta = -x*del*sin(theta)+y*del*cos(theta);
                  kernel.at<float>(hks+y,hks+x) = (float)exp(-0.5*(pow(x_theta,2)+pow(y_theta,2))/pow(sigma,2))* cos(2*CV_PI*x_theta/lmbd + psi);
              }
          }
          return kernel;
      }

 int Process(int ks,int sigma, int lamda,  int psi)
     {
	 seg=abs(grey);
	 for (int i=0;i<12;i++)
     	{
     		cv::Mat kernel = mkKernel(ks, sigma, i*30, lamda, psi);
     		cv::filter2D(grey, dest[i], CV_32F, kernel);
     		seg+=abs(dest[i]);
     	}
     	return 0;
     }
 void detect_and_draw(IplImage* img)
 {
	 static CvMemStorage* storage = 0;
	 static CvHaarClassifierCascade* cascade = 0;
	 int scale = 1;
	 IplImage* temp = cvCreateImage(cvSize(img->width / scale, img->height / scale), 8, 3);
	 CvPoint pt1, pt2;
 	int i;
 	cascade = (CvHaarClassifierCascade*)cvLoad(cascade_name, 0, 0, 0);
if (!cascade)
 	{
 		fprintf(stderr, "ERROR: Could not load classifier cascade\n");
 		return;
 	}
 	storage = cvCreateMemStorage(0);
 	cvClearMemStorage(storage);
if (cascade)
 	{
 		CvSeq* faces = cvHaarDetectObjects(img, cascade, storage,
 			1.1, 2, CV_HAAR_DO_CANNY_PRUNING,
 			cvSize(40, 40));
 		for (i = 0; i < (faces ? faces->total : 0); i++)
 		{
 			CvRect* r = (CvRect*)cvGetSeqElem(faces, i);
 			pt1.x = r->x*scale;
 			pt2.x = (r->x + r->width)*scale;
 			pt1.y = r->y*scale;
 			pt2.y = (r->y + r->height)*scale;
 			cvRectangle(img, pt1, pt2, CV_RGB(255, 0, 0), 3, 8, 0);
 		}
 	}
 	cvShowImage("result", img);
 	cvReleaseImage(&temp);
 }

int main()
{

CvCapture *camera=cvCaptureFromFile("sample6.avi");
cvNamedWindow("img");
while (cvWaitKey(10)!=atoi("q"))
{
    double t1=(double)cvGetTickCount();
    IplImage *imge=cvQueryFrame(camera);
    cv::Mat B1 = cvarrToMat(imge);
    //IplImage * image = &((IplImage)B1);
    cvShowImage("img",imge);
    //cv::imshow("img1",B1);
    cv::cvtColor(B1, src, CV_BGR2GRAY);
    //cv::imshow("img1",src);
    src.convertTo(grey, CV_32F, 1.0/255, 0);
    //cv::imshow("img1",grey);
    Process(21, 5, 50,90);
    seg= seg-abs(grey);
    //seg/=4;
    cv::imshow("seg",seg);



    		 IplImage copy = seg;
    		 IplImage* new_image = &copy;

    		IplImage* haar = cvCreateImage(cvGetSize(new_image), IPL_DEPTH_8U, 1);


    		detect_and_draw(haar);

    seg*=0;

}
cvReleaseCapture(&camera);
cvDestroyWindow("result");
}
