#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <math.h>
#include<iostream>
#include<stdio.h>

    cv::Mat mkKernel(int ks, double sig, double th, double lm, double ps)
    {
        int hks = (ks-1)/2;
        double theta = th*CV_PI/180;
        double psi = ps*CV_PI/180;
        double del = 2.0/(ks-1);
        double lmbd = lm;
        double sigma = sig/ks;
        double x_theta;
        double y_theta;
        cv::Mat kernel(ks,ks, CV_32F);
        for (int y=-hks; y<=hks; y++)
        {
            for (int x=-hks; x<=hks; x++)
            {
                x_theta = x*del*cos(theta)+y*del*sin(theta);
                y_theta = -x*del*sin(theta)+y*del*cos(theta);
                kernel.at<float>(hks+y,hks+x) = (float)exp(-0.5*(pow(x_theta,2)+pow(y_theta,2))/pow(sigma,2))* cos(2*CV_PI*x_theta/lmbd + psi);
            }
        }
        return kernel;
    }

    cv::Mat src_f;
    cv::Mat dest[12];

    int Process(int ks,int sigma, int lamda,  int psi)
    {
    	char buffer [50],name[30];
    	cv::Mat seg=abs(src_f);

    	for (int i=0;i<12;i++)
    	{
    		cv::Mat kernel = mkKernel(ks, sigma, i*30, lamda, psi);
    		cv::filter2D(src_f, dest[i], CV_32F, kernel);
    		sprintf (buffer, "angle of %d", i*30);
    		cv::imshow(buffer, dest[i]);
    		seg+=abs(dest[i]);



    	}

    	seg= seg-src_f;
    	seg/=4;
    	cv::imshow("segm",seg);


        return 0;
    }

    int main(int argc, char** argv)
    {
    	int ks=21;
    	cv::Mat image;
    	image=cv::imread("lena.jpg",1);


        while(true)
        {

        	        cv::imshow("Src", image);
        	        cv::Mat src;
        	        cv::cvtColor(image, src, CV_BGR2GRAY);
        	        src.convertTo(src_f, CV_32F, 1.0/255, 0);
        	        if (!ks%2)
        	        {
        	            ks+=1;
        	        }

        	        Process(21, 5, 50,90);

        	        cv::waitKey(0);

        }

        return 0;
    }
