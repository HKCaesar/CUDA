#include "hip/hip_runtime.h"
#include "opencv2/core/core.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/video/background_segm.hpp"
#include "opencv2/highgui/highgui.hpp"
#include <stdio.h>
#include <iostream>
#include <vector>
#include <math.h>
#include <opencv2/core/core.hpp>

using namespace std;
using namespace cv;

cv::Mat seg[3];
cv::Mat src;
cv::Mat grey;
cv::Mat dest[3];


const char* keys =
{
    "{c |camera |true | use camera or not}"
    "{fn|file_name|tree.avi | movie file }"
};

cv::Mat mkKernel(int ks, double sig, double th, double lm, double ps)
      {
          int hks = (ks-1)/2;
          double theta = th*CV_PI/180;
          double psi = ps*CV_PI/180;
          double del = 2.0/(ks-1);
          double lmbd = lm;
          double sigma = sig/ks;
          double x_theta;
          double y_theta;
          cv::Mat kernel(ks,ks, CV_32F);
          for (int y=-hks; y<=hks; y++)
          {
              for (int x=-hks; x<=hks; x++)
              {
                  x_theta = x*del*cos(theta)+y*del*sin(theta);
                  y_theta = -x*del*sin(theta)+y*del*cos(theta);
                  kernel.at<float>(hks+y,hks+x) = (float)exp(-0.5*(pow(x_theta,2)+pow(y_theta,2))/pow(sigma,2))* cos(2*CV_PI*x_theta/lmbd + psi);
              }
          }
          return kernel;
      }

 int Process(int ks,int sigma, int lamda, int psi)
     {
seg[0]=abs(grey);
seg[1]=abs(grey);
seg[2]=abs(grey);

for (int i=0;i<3;i++)
      {
      cv::Mat kernel = mkKernel(ks, sigma, i*120, lamda, psi);
      cv::filter2D(grey, dest[i], CV_32F, kernel);
      seg[i]+=abs(dest[i]);
      }
      return 0;
     }

//this is a sample for foreground detection functions
int main(int argc, const char** argv)
{
    CvCapture *camera=cvCaptureFromFile("sample.avi");
    bool update_bg_model = true;
    namedWindow("real", WINDOW_NORMAL);

    BackgroundSubtractorMOG2 bg_model,bg_model1;//(100, 3, 0.3, 5);

    Mat img1,img, fgmask, fgimg, B1,fgmask1,fgimg1;

    for(;;)
    {
     IplImage *img_c=cvQueryFrame(camera);
     cv::Mat img1 = cvarrToMat(img_c);

        if( img1.empty() )
            break;
        imshow("real",img1);
        cvtColor(img1, src, CV_BGR2GRAY);
        src.convertTo(grey, CV_32F, 1.0/255, 0);
        //imshow("grey",grey);
        Process(21, 5, 50,90);

        seg[0]= seg[0]-abs(grey);
        seg[1]= seg[1]-abs(grey);
        seg[2]= seg[2]-abs(grey);
        Mat segmn=seg[0]+seg[1]+seg[2];
        imshow("only gabor(3 angles)",segmn);

        int m=256;
        seg[0]*= m;
        seg[1]*= m;
        seg[2]*= m;

        //imshow("seg0",seg[0]);

        std::vector<cv::Mat> images(3);
        images.at(0) = seg[0]; //for blue channel
        images.at(1) = seg[1]; //for green channel
        images.at(2) = seg[2]; //for red channel

        //cv::Mat colorImage;
        cv::merge(images, img);
        //imshow("colorimage",colorImage);
        if( fgimg.empty() )
          fgimg.create(img.size(), img.type());
        //update the model
        bg_model(img, fgmask, update_bg_model ? -1 : 0);

        fgimg = Scalar::all(0);
        img.copyTo(fgimg, fgmask);

        Mat bgimg;
        bg_model.getBackgroundImage(bgimg);
        imshow("GMM and Gabor fgmask", fgmask);
        imshow("GMM and gabor fgimg", fgimg);

        //cv::merge(images, img);
               //imshow("colorimage",colorImage);
               if( fgimg1.empty() )
                 fgimg1.create(img1.size(), img1.type());
               //update the model
               bg_model1(img1, fgmask1, update_bg_model ? -1 : 0);

               fgimg1 = Scalar::all(0);
               img1.copyTo(fgimg1, fgmask1);

               Mat bgimg1;
               bg_model1.getBackgroundImage(bgimg1);
               imshow("only GMM foreground mask", fgmask1);
               imshow("only GMM foreground image", fgimg1);
        char k = (char)waitKey(30);
        if( k == 27 ) break;
        if( k == ' ' )
        {
            update_bg_model = !update_bg_model;
            if(update_bg_model)
                printf("Background update is on\n");
            else
                printf("Background update is off\n");
        }
    }

    return 0;
}
